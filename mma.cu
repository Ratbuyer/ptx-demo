#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <assert.h>
#include <hip/hip_fp16.h>
#include <random>
#include <iostream>

void cpu_matmul_row_major(half *A, half *B, float *C, int M, int N, int K)
{
  // CPU matmul
  for (int i = 0; i < M; i++)
  {
    for (int j = 0; j < N; j++)
    {
      float sum = 0.0f;
      for (int k = 0; k < K; k++)
      {
        sum += __half2float(A[i * K + k]) * __half2float(B[k * N + j]);
      }
      C[i * N + j] = sum;
    }
  }
}

void compare_matrix(float *h_C, float *CPU_C, int length)
{
  const int verbose = 0;

  for (int i = 0; i < length; i++)
  {
    if (CPU_C[i] != h_C[i] && (verbose == 0))
      printf("incorrect: %f, C: %f\n", CPU_C[i], h_C[i]);

    if (verbose)
      printf("incorrect: %f, C: %f\n", CPU_C[i], h_C[i]);
  }
}

__global__ void fill_B(half *ptr, int offset)
{

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  int base = offset * y + 4 * x;

  ptr[base] = 1.0f * (x % 2);
  ptr[base + 1] = 1.0f * (y % 3);
  ptr[base + 2] = 1.0f * (x % 4);
  ptr[base + 3] = 1.0f * (y % 5);
}

__global__ void fill_A(half *ptr, int offset)
{
  const char *pattern = "0110";

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  int base = offset * y + 4 * x;

  for (int i = 0; i < 4; i++)
  {
    if (pattern[i] == '1')
      ptr[base + i] = 1.0f;
    else
      ptr[base + i] = 0.0f;
  }
}

__device__ __forceinline__ unsigned int merge_half2_to_b32(half2 values)
{
  //===========merge two half into one .b32 register====================

  unsigned int merged_value;
  unsigned short *value_ptr = reinterpret_cast<unsigned short *>(&values);
  unsigned int upper_half = static_cast<unsigned int>(value_ptr[0]);
  unsigned int lower_half = static_cast<unsigned int>(value_ptr[1]);

  merged_value = (upper_half << 16) | lower_half;
  return merged_value;
}

__global__ void compress_A(half *ptr, int M, int K, unsigned *compressed_A)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  half2 values;
  reinterpret_cast<half&>(values.x) = ptr[y * K + 2 * x];
  reinterpret_cast<half&>(values.y) = ptr[y * K + 2 * x + 1];
  compressed_A[y * K / 2 + x] = merge_half2_to_b32(values);
}

__global__ void compress_B(half *ptr, int K, int N, unsigned *compressed_B)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  half2 values;
  reinterpret_cast<half&>(values.x) = ptr[2 * y * N + x];
  reinterpret_cast<half&>(values.y) = ptr[(2 * y + 1) * N + x];
  compressed_B[y * N + x] = merge_half2_to_b32(values);
}

__global__ void kernel(int M, int N, int K,
                       unsigned *A, unsigned *B, float *C)
{

  // Each Warp calculates one 16x8 tile in C, one tile of C is calculated from
  // one 16x16 tile of A multiple by one 16x8 tile of B

  // Each thread in a warp holds 8 values of A,
  // 4 values of B and stores 4 values of C

  using namespace nvcuda;
  //===================thread information================
  int x = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  int laneid = threadIdx.x;
  int groupID = laneid >> 2;
  int threadID_in_group = laneid % 4;
  //====================do accumulation in shared memory=========================
  __shared__ float tile[16 * 8];

  tile[groupID * 8 + threadID_in_group * 2] = 0;
  tile[groupID * 8 + threadID_in_group * 2 + 1] = 0;
  tile[(groupID + 8) * 8 + threadID_in_group * 2] = 0;
  tile[(groupID + 8) * 8 + threadID_in_group * 2 + 1] = 0;
  //==========================variable declaration===========================
  int A_row = x * 16,
      B_row,
      A_col,
      B_col = y * 8,
      A_base, B_base, C_base; // the start address of each tile

  C_base = B_col + A_row * N;

  unsigned A1, A2, A3, A4, B1, B2;
  //================================================================
  for (int k = 0; k < K; k += 16)
  { // iterate the reduction axis
    A_col = k;
    B_row = k;

    //========================start address of each tile=================
    A_base = A_col / 2 + A_row * K / 2;
    B_base = B_col + B_row * N / 2;
    //========================load 8 values of A======================================
    A1 = A[A_base + groupID * K / 2 + threadID_in_group];
    A2 = A[A_base + (groupID + 8) * K / 2 + threadID_in_group];
    A3 = A[A_base + groupID * K / 2 + threadID_in_group + 4];
    A4 = A[A_base + (groupID + 8) * K / 2 + threadID_in_group + 4];
    //========================load 4 values of B===========================
    B1 = B[B_base + threadID_in_group * N + groupID];
    B2 = B[B_base + (threadID_in_group + 4) * N + groupID];
    //=======================mma instruction which does matmul========================
    asm volatile(
        "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
        "{%0,  %1,  %2,  %3},"
        "{%4,  %5,  %6,  %7},"
        "{%8,  %9},"
        "{%10, %11, %12, %13};"
        : "=f"(tile[groupID * 8 + threadID_in_group * 2]),
          "=f"(tile[groupID * 8 + threadID_in_group * 2 + 1]),
          "=f"(tile[(groupID + 8) * 8 + threadID_in_group * 2]),
          "=f"(tile[(groupID + 8) * 8 + threadID_in_group * 2 + 1])
        : "r"(A1), "r"(A2), "r"(A3), "r"(A4),
          "r"(B1), "r"(B2),
          "f"(tile[groupID * 8 + threadID_in_group * 2]),
          "f"(tile[groupID * 8 + threadID_in_group * 2 + 1]),
          "f"(tile[(groupID + 8) * 8 + threadID_in_group * 2]),
          "f"(tile[(groupID + 8) * 8 + threadID_in_group * 2 + 1]));
    //========================================================================
  }

  C[C_base + groupID * N + threadID_in_group * 2] =
      tile[groupID * 8 + threadID_in_group * 2];
  C[C_base + groupID * N + threadID_in_group * 2 + 1] =
      tile[groupID * 8 + threadID_in_group * 2 + 1];
  C[C_base + (groupID + 8) * N + threadID_in_group * 2] =
      tile[(groupID + 8) * 8 + threadID_in_group * 2];
  C[C_base + (groupID + 8) * N + threadID_in_group * 2 + 1] =
      tile[(groupID + 8) * 8 + threadID_in_group * 2 + 1];
}

void matmul(int m, int n, int k, unsigned *A, unsigned *B, float *C)
{
  kernel<<<dim3(m / 16, n / 8), dim3(32, 1)>>>(m, n, k, A, B, C);
  hipDeviceSynchronize();
}

int main(int argc, char *argv[])
{
  assert(argc == 5);
  //=============================allocate host================================
  const int M = atoi(argv[1]);
  const int N = atoi(argv[2]);
  const int K = atoi(argv[3]);

  assert(M % 16 == 0);
  assert(N % 16 == 0);
  assert(K % 32 == 0);

  half *h_A, *h_B;
  float *h_C, *CPU_C;

  h_A = (half *)malloc(M * K * sizeof(half));
  h_B = (half *)malloc(K * N * sizeof(half));
  h_C = (float *)malloc(M * N * sizeof(float));
  CPU_C = (float *)malloc(M * N * sizeof(float));
  //==============================allocate device========================================
  half *d_A, *d_B;
  float *d_C;
  unsigned *compressed_A, *compressed_B;

  hipMalloc((void **)&d_A, M * K * sizeof(half));
  hipMalloc((void **)&d_B, K * N * sizeof(half));
  hipMalloc((void **)&d_C, M * N * sizeof(float));
  hipMalloc((void **)&compressed_A, M * K / 2 * sizeof(unsigned));
  hipMalloc((void **)&compressed_B, K / 2 * N * sizeof(unsigned));
  //==============================initialize matrices======================================
  fill_A<<<dim3(K / (16 * 4), M / 16), dim3(16, 16)>>>(d_A, K);
  fill_B<<<dim3(N / (16 * 4), K / 16), dim3(16, 16)>>>(d_B, N);
  //============================compressing A, B===============================
  int iter = atoi(argv[4]);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  float milliseconds = 0;

  hipEventRecord(start);

  for (int i = 0; i < iter; i++)
  {
    compress_A<<<dim3(K / (16 * 2), M / 16), dim3(16, 16)>>>(d_A, M, K, compressed_A);
    compress_B<<<dim3(N / 16, K / (16 * 2)), dim3(16, 16)>>>(d_B, K, N, compressed_B);
  }

  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&milliseconds, start, end);
  std::cout << "compression time: "
            << milliseconds / iter
            << " ms"
            << " averaged accross "
            << iter
            << " iterations "
            << std::endl;
  //=============================profile matmul===============================

  hipEventRecord(start);

  for (int i = 0; i < iter; i++)
  {
    matmul(M, N, K, compressed_A, compressed_B, d_C);
  }

  hipEventRecord(end);
  hipEventSynchronize(end);

  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, end);
  std::cout << "Elapsed time: "
            << milliseconds / iter
            << " ms"
            << " averaged accross "
            << iter
            << " iterations "
            << std::endl;
  //===============================checking results================================
  if (1)
  {
    printf("Checking\n");
    hipMemcpy(h_A, d_A, M * K * sizeof(half), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, K * N * sizeof(half), hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    cpu_matmul_row_major(h_A, h_B, CPU_C, M, N, K);
    compare_matrix(h_C, CPU_C, M * N);
  }
  //============================free allocations==================================
  free(h_A);
  free(h_B);
  free(h_C);
  free(CPU_C);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(compressed_A);
  hipFree(compressed_B);
  //===================================return==========================
  printf("COMPLETED!\n");
  return 0;
}